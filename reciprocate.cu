
#include <hip/hip_runtime.h>
extern "C" {
	__global__ void reciprocate(float *a, int n)
	{
		int i;

		for (i = 0; i < n; ++i)
			a[blockIdx.x] = 1. / a[blockIdx.x];
	}
}
