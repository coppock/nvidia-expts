
#include <hip/hip_runtime.h>
extern "C" {
	__global__ void reciprocate(float *a, int n)
	{
		int i;

		for (i = 0; i < n; ++i)
			a[0] = 1. / a[0];
	}
}
