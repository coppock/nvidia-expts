
#include <hip/hip_runtime.h>
extern "C" {
	__global__ void read_clock(clock_t *stamps, size_t n)
	{
		size_t i;

		for (i = 0; i < n; ++i)
			stamps[i] = clock();
	}
}
